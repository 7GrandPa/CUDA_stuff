#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <cmath>
#include <iostream>

hipError_t hipMalloc(void** devPTr, size_t size);
hipError_t hipMemcpy(void* dst, const void* src, size_t count, hipMemcpyKind kind);

__global__ void add(float *x, float *y, float *z, int n)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride) {
		z[i] = x[i] + y[i];
	}   
}

int main()
{
	int N = 1 << 20;
	int nBytes = N * sizeof(float);

	// apply for host memory
	float *x, *y, *z;
	x = (float*)malloc(nBytes);
	y = (float*)malloc(nBytes);
	z = (float*)malloc(nBytes);

	// init the data
	for (int i=0; i < N; i++) {
		x[i] = 10.0;
		y[i] = 20.0;
	}

	// apply for device mem
	float *d_x, *d_y, *d_z;
	hipMalloc((void**)&d_x, nBytes);
	hipMalloc((void**)&d_y, nBytes);
	hipMalloc((void**)&d_z, nBytes);

	// copy data from host to device
	hipMemcpy((void*)d_x, (void*)x, nBytes, hipMemcpyHostToDevice);
	hipMemcpy((void*)d_y, (void*)y, nBytes, hipMemcpyHostToDevice);

	// define the kernel configuration
	dim3 blockSize(256);
	dim3 gridSize((N + blockSize.x - 1) / blockSize.x);

	//execute
	add << < gridSize, blockSize >> > (d_x, d_y, d_z, N);

	//copy data from gpu to host
	hipMemcpy((void*)z, (void*)d_z, nBytes, hipMemcpyHostToDevice);

	//
	float maxError = 0.0;
	for (int i = 0; i < N; i++)
		maxError = fmax(maxError, fabs(z[i] - 30.0));
	std::cout << "������: " << maxError << std::endl;

	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_z);

	//free host mem
	free(x);
	free(y);
	free(z);

	return 0;
}


